#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <math.h>
#include "canny.h"

#define _USE_MATH_DEFINES
#define RGB2GRAY_CONST_ARR_SIZE 3
#define STRONG_EDGE 0xFFFF
#define NON_EDGE 0x0

//*****************************************************************************************
// CUDA Gaussian Filter Implementation
//*****************************************************************************************

///
/// \brief Apply gaussian filter. This is the CUDA kernel for applying a gaussian blur to an image.
///
__global__
void cu_apply_gaussian_filter(pixel_t* in_pixels, pixel_t* out_pixels, int rows, int cols, double* in_kernel)
{
    //copy kernel array from global memory to a shared array
    __shared__ double kernel[KERNEL_SIZE][KERNEL_SIZE];
    for (int i = 0; i < KERNEL_SIZE; ++i) {
        for (int j = 0; j < KERNEL_SIZE; ++j) {
            kernel[i][j] = in_kernel[i * KERNEL_SIZE + j];
        }
    }

    __syncthreads();

    //determine id of thread which corresponds to an individual pixel
    int pixNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (pixNum >= 0 && pixNum < rows * cols) {

        double kernelSum;
        double redPixelVal;
        double greenPixelVal;
        double bluePixelVal;

        //Apply Kernel to each pixel of image
        for (int i = 0; i < KERNEL_SIZE; ++i) {
            for (int j = 0; j < KERNEL_SIZE; ++j) {

                //check edge cases, if within bounds, apply filter
                if (((pixNum + ((i - ((KERNEL_SIZE - 1) / 2)) * cols) + j - ((KERNEL_SIZE - 1) / 2)) >= 0)
                    && ((pixNum + ((i - ((KERNEL_SIZE - 1) / 2)) * cols) + j - ((KERNEL_SIZE - 1) / 2)) <= rows * cols - 1)
                    && (((pixNum % cols) + j - ((KERNEL_SIZE - 1) / 2)) >= 0)
                    && (((pixNum % cols) + j - ((KERNEL_SIZE - 1) / 2)) <= (cols - 1))) {

                    redPixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2)) * cols) + j - ((KERNEL_SIZE - 1) / 2)].red;
                    greenPixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2)) * cols) + j - ((KERNEL_SIZE - 1) / 2)].green;
                    bluePixelVal += kernel[i][j] * in_pixels[pixNum + ((i - ((KERNEL_SIZE - 1) / 2)) * cols) + j - ((KERNEL_SIZE - 1) / 2)].blue;
                    kernelSum += kernel[i][j];
                }
            }
        }

        //update output image
        out_pixels[pixNum].red = redPixelVal / kernelSum;
        out_pixels[pixNum].green = greenPixelVal / kernelSum;
        out_pixels[pixNum].blue = bluePixelVal / kernelSum;
    }
}

//*****************************************************************************************
// CUDA Intensity Gradient Implementation
//*****************************************************************************************

///
/// \brief Compute gradient (first order derivative x and y). This is the CUDA kernel for taking the derivative of color contrasts in adjacent images.
///
__global__
void cu_compute_intensity_gradient(pixel_t* in_pixels, pixel_channel_t_signed* deltaX_channel, pixel_channel_t_signed* deltaY_channel, unsigned parser_length, unsigned offset)
{
    // compute delta X ***************************
    // deltaX = f(x+1) - f(x-1)

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    /* condition here skips first and last row */
    if ((idx > offset) && (idx < (parser_length * offset) - offset))
    {
        int16_t deltaXred = 0;
        int16_t deltaYred = 0;
        int16_t deltaXgreen = 0;
        int16_t deltaYgreen = 0;
        int16_t deltaXblue = 0;
        int16_t deltaYblue = 0;

        /* first column */
        if ((idx % offset) == 0)
        {
            // gradient at the first pixel of each line
            // note: at the edge pix[idx-1] does NOT exist
            deltaXred = (int16_t)(in_pixels[idx + 1].red - in_pixels[idx].red);
            deltaXgreen = (int16_t)(in_pixels[idx + 1].green - in_pixels[idx].green);
            deltaXblue = (int16_t)(in_pixels[idx + 1].blue - in_pixels[idx].blue);
            // gradient at the first pixel of each line
            // note: at the edge pix[idx-1] does NOT exist
            deltaYred = (int16_t)(in_pixels[idx + offset].red - in_pixels[idx].red);
            deltaYgreen = (int16_t)(in_pixels[idx + offset].green - in_pixels[idx].green);
            deltaYblue = (int16_t)(in_pixels[idx + offset].blue - in_pixels[idx].blue);
        }
        /* last column */
        else if ((idx % offset) == (offset - 1))
        {
            deltaXred = (int16_t)(in_pixels[idx].red - in_pixels[idx - 1].red);
            deltaXgreen = (int16_t)(in_pixels[idx].green - in_pixels[idx - 1].green);
            deltaXblue = (int16_t)(in_pixels[idx].blue - in_pixels[idx - 1].blue);
            deltaYred = (int16_t)(in_pixels[idx].red - in_pixels[idx - offset].red);
            deltaYgreen = (int16_t)(in_pixels[idx].green - in_pixels[idx - offset].green);
            deltaYblue = (int16_t)(in_pixels[idx].blue - in_pixels[idx - offset].blue);
        }
        /* gradients where NOT edge */
        else
        {
            deltaXred = (int16_t)(in_pixels[idx + 1].red - in_pixels[idx - 1].red);
            deltaXgreen = (int16_t)(in_pixels[idx + 1].green - in_pixels[idx - 1].green);
            deltaXblue = (int16_t)(in_pixels[idx + 1].blue - in_pixels[idx - 1].blue);
            deltaYred = (int16_t)(in_pixels[idx + offset].red - in_pixels[idx - offset].red);
            deltaYgreen = (int16_t)(in_pixels[idx + offset].green - in_pixels[idx - offset].green);
            deltaYblue = (int16_t)(in_pixels[idx + offset].blue - in_pixels[idx - offset].blue);
        }
        deltaX_channel[idx] = (int16_t)(0.2989 * deltaXred + 0.5870 * deltaXgreen + 0.1140 * deltaXblue);
        deltaY_channel[idx] = (int16_t)(0.2989 * deltaYred + 0.5870 * deltaYgreen + 0.1140 * deltaYblue);
    }
}

//*****************************************************************************************
// CUDA Gradient Magnitude Implementation
//*****************************************************************************************

///
/// \brief Compute magnitude of gradient(deltaX & deltaY) per pixel.
///
__global__
void cu_magnitude(pixel_channel_t_signed* deltaX, pixel_channel_t_signed* deltaY, pixel_channel_t* out_pixel, unsigned parser_length, unsigned offset)
{
    //computation
    //Assigned a thread to each pixel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 0 && idx < parser_length * offset) {
        out_pixel[idx] = (pixel_channel_t)(sqrt((double)deltaX[idx] * deltaX[idx] +
            (double)deltaY[idx] * deltaY[idx]) + 0.5);
    }
}

//*****************************************************************************************
// CUDA Non Maximal Suppression Implementation
//*****************************************************************************************

///
/// \brief Non Maximal Suppression
/// If the centre pixel is not greater than neighboured pixels in the direction,
/// then the center pixel is set to zero.
/// This process results in one pixel wide ridges.
///
__global__
void cu_suppress_non_max(pixel_channel_t* mag, pixel_channel_t_signed* deltaX, pixel_channel_t_signed* deltaY, pixel_channel_t* nms, unsigned parser_length, unsigned offset)
{

    const pixel_channel_t SUPPRESSED = 0;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 0 && idx < parser_length * offset)
    {
        float alpha;
        float mag1, mag2;
        // put zero all boundaries of image
        // TOP edge line of the image
        if ((idx >= 0) && (idx < offset))
            nms[idx] = 0;

        // BOTTOM edge line of image
        else if ((idx >= (parser_length - 1) * offset) && (idx < (offset * parser_length)))
            nms[idx] = 0;

        // LEFT & RIGHT edge line
        else if (((idx % offset) == 0) || ((idx % offset) == (offset - 1)))
        {
            nms[idx] = 0;
        }

        else // not the boundaries
        {
            // if magnitude = 0, no edge
            if (mag[idx] == 0)
                nms[idx] = SUPPRESSED;
            else {
                if (deltaX[idx] >= 0)
                {
                    if (deltaY[idx] >= 0)  // dx >= 0, dy >= 0
                    {
                        if ((deltaX[idx] - deltaY[idx]) >= 0)       // direction 1 (SEE, South-East-East)
                        {
                            alpha = (float)deltaY[idx] / deltaX[idx];
                            mag1 = (1 - alpha) * mag[idx + 1] + alpha * mag[idx + offset + 1];
                            mag2 = (1 - alpha) * mag[idx - 1] + alpha * mag[idx - offset - 1];
                        }
                        else                                // direction 2 (SSE)
                        {
                            alpha = (float)deltaX[idx] / deltaY[idx];
                            mag1 = (1 - alpha) * mag[idx + offset] + alpha * mag[idx + offset + 1];
                            mag2 = (1 - alpha) * mag[idx - offset] + alpha * mag[idx - offset - 1];
                        }
                    }
                    else  // dx >= 0, dy < 0
                    {
                        if ((deltaX[idx] + deltaY[idx]) >= 0)    // direction 8 (NEE)
                        {
                            alpha = (float)-deltaY[idx] / deltaX[idx];
                            mag1 = (1 - alpha) * mag[idx + 1] + alpha * mag[idx - offset + 1];
                            mag2 = (1 - alpha) * mag[idx - 1] + alpha * mag[idx + offset - 1];
                        }
                        else                                // direction 7 (NNE)
                        {
                            alpha = (float)deltaX[idx] / -deltaY[idx];
                            mag1 = (1 - alpha) * mag[idx + offset] + alpha * mag[idx + offset - 1];
                            mag2 = (1 - alpha) * mag[idx - offset] + alpha * mag[idx - offset + 1];
                        }
                    }
                }

                else
                {
                    if (deltaY[idx] >= 0) // dx < 0, dy >= 0
                    {
                        if ((deltaX[idx] + deltaY[idx]) >= 0)    // direction 3 (SSW)
                        {
                            alpha = (float)-deltaX[idx] / deltaY[idx];
                            mag1 = (1 - alpha) * mag[idx + offset] + alpha * mag[idx + offset - 1];
                            mag2 = (1 - alpha) * mag[idx - offset] + alpha * mag[idx - offset + 1];
                        }
                        else                                // direction 4 (SWW)
                        {
                            alpha = (float)deltaY[idx] / -deltaX[idx];
                            mag1 = (1 - alpha) * mag[idx - 1] + alpha * mag[idx + offset - 1];
                            mag2 = (1 - alpha) * mag[idx + 1] + alpha * mag[idx - offset + 1];
                        }
                    }

                    else // dx < 0, dy < 0
                    {
                        if ((-deltaX[idx] + deltaY[idx]) >= 0)   // direction 5 (NWW)
                        {
                            alpha = (float)deltaY[idx] / deltaX[idx];
                            mag1 = (1 - alpha) * mag[idx - 1] + alpha * mag[idx - offset - 1];
                            mag2 = (1 - alpha) * mag[idx + 1] + alpha * mag[idx + offset + 1];
                        }
                        else                                // direction 6 (NNW)
                        {
                            alpha = (float)deltaX[idx] / deltaY[idx];
                            mag1 = (1 - alpha) * mag[idx - offset] + alpha * mag[idx - offset - 1];
                            mag2 = (1 - alpha) * mag[idx + offset] + alpha * mag[idx + offset + 1];
                        }
                    }
                }

                // non-maximal suppression
                // compare mag1, mag2 and mag[t]
                // if mag[t] is smaller than one of the neighbours then suppress it
                if ((mag[idx] < mag1) || (mag[idx] < mag2))
                    nms[idx] = SUPPRESSED;
                else
                {
                    nms[idx] = mag[idx];
                }

            } // END OF ELSE (mag != 0)
        } // END OF FOR(j)
    } // END OF FOR(i)
}

//*****************************************************************************************
// CUDA Hysteresis Implementation
//*****************************************************************************************

///
/// \brief This is a helper function that runs on the GPU.
///
/// It checks if the eight immediate neighbors of a pixel at a given index are above
/// a low threshold, and if they are, sets them to strong edges. This effectively
/// connects the edges.
///
__device__
void trace_immed_neighbors(pixel_channel_t* out_pixels, pixel_channel_t* in_pixels,
    unsigned idx, pixel_channel_t t_low, unsigned img_width)
{
    /* directions representing indices of neighbors */
    unsigned n, s, e, w;
    unsigned nw, ne, sw, se;

    /* get indices */
    n = idx - img_width;
    nw = n - 1;
    ne = n + 1;
    s = idx + img_width;
    sw = s - 1;
    se = s + 1;
    w = idx - 1;
    e = idx + 1;

    if (in_pixels[nw] >= t_low) {
        out_pixels[nw] = STRONG_EDGE;
    }
    if (in_pixels[n] >= t_low) {
        out_pixels[n] = STRONG_EDGE;
    }
    if (in_pixels[ne] >= t_low) {
        out_pixels[ne] = STRONG_EDGE;
    }
    if (in_pixels[w] >= t_low) {
        out_pixels[w] = STRONG_EDGE;
    }
    if (in_pixels[e] >= t_low) {
        out_pixels[e] = STRONG_EDGE;
    }
    if (in_pixels[sw] >= t_low) {
        out_pixels[sw] = STRONG_EDGE;
    }
    if (in_pixels[s] >= t_low) {
        out_pixels[s] = STRONG_EDGE;
    }
    if (in_pixels[se] >= t_low) {
        out_pixels[se] = STRONG_EDGE;
    }
}

///
/// \brief CUDA implementation of Canny hysteresis high thresholding.
///
/// This kernel is the first pass in the parallel hysteresis step.
/// It launches a thread for every pixel and checks if the value of that pixel
/// is above a high threshold. If it is, the thread marks it as a strong edge (set to 1)
/// in a pixel map and sets the value to the channel max. If it is not, the thread sets
/// the pixel map at the index to 0 and zeros the output buffer space at that index.
///
/// The output of this step is a mask of strong edges and an output buffer with white values
/// at the mask indices which are set.
///
__global__
void cu_hysteresis_high(pixel_channel_t* out_pixels, pixel_channel_t* in_pixels, unsigned* strong_edge_mask,
    pixel_channel_t t_high, unsigned img_height, unsigned img_width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (img_height * img_width)) {
        /* apply high threshold */
        if (in_pixels[idx] > t_high) {
            strong_edge_mask[idx] = 1;
            out_pixels[idx] = STRONG_EDGE;
        }
        else {
            strong_edge_mask[idx] = 0;
            out_pixels[idx] = NON_EDGE;
        }
    }
}

///
/// \brief CUDA implementation of Canny hysteresis low thresholding.
///
/// This kernel is the second pass in the parallel hysteresis step. 
/// It launches a thread for every pixel, but skips the first and last rows and columns.
/// For surviving threads, the pixel at the thread ID index is checked to see if it was 
/// previously marked as a strong edge in the first pass. If it was, the thread checks 
/// their eight immediate neighbors and connects them (marks them as strong edges)
/// if the neighbor is above the low threshold.
///
/// The output of this step is an output buffer with both "strong" and "connected" edges
/// set to whtie values. This is the final edge detected image.
///
__global__
void cu_hysteresis_low(pixel_channel_t* out_pixels, pixel_channel_t* in_pixels, unsigned* strong_edge_mask,
    unsigned t_low, unsigned img_height, unsigned img_width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ((idx > img_width)                               /* skip first row */
        && (idx < (img_height * img_width) - img_width) /* skip last row */
        && ((idx % img_width) < (img_width - 1))        /* skip last column */
        && ((idx % img_width) > (0)))                  /* skip first column */
    {
        if (1 == strong_edge_mask[idx]) { /* if this pixel was previously found to be a strong edge */
            trace_immed_neighbors(out_pixels, in_pixels, idx, t_low, img_width);
        }
    }
}

//*****************************************************************************************
// Test/Debug hooks for separate kernels
// These generally aren't to be used, but can serve as drop-in replacements for any
// particular step of the algorithm's serial implementation.
// Useful for debugging individual kernels.
//*****************************************************************************************

void cu_test_gradient(pixel_t* buf0, pixel_channel_t_signed* deltaX_gray, pixel_channel_t_signed* deltaY_gray, unsigned rows, unsigned cols)
{
    pixel_t* in_pixels;
    pixel_channel_t_signed* deltaX;
    pixel_channel_t_signed* deltaY;

    hipMalloc((void**)&in_pixels, sizeof(pixel_t) * rows * cols);
    hipMalloc((void**)&deltaX, sizeof(pixel_channel_t_signed) * rows * cols);
    hipMalloc((void**)&deltaY, sizeof(pixel_channel_t_signed) * rows * cols);

    hipMemcpy(in_pixels, buf0, rows * cols * sizeof(pixel_t), hipMemcpyHostToDevice);

    cu_compute_intensity_gradient << <(rows * cols) / 1024, 1024 >> > (in_pixels, deltaX, deltaY, rows, cols);

    hipMemcpy(deltaX_gray, deltaX, rows * cols * sizeof(pixel_channel_t_signed), hipMemcpyDeviceToHost);
    hipMemcpy(deltaY_gray, deltaY, rows * cols * sizeof(pixel_channel_t_signed), hipMemcpyDeviceToHost);

    hipFree(in_pixels);
    hipFree(deltaX);
    hipFree(deltaY);
}

void cu_test_mag(pixel_channel_t_signed* deltaX, pixel_channel_t_signed* deltaY, pixel_channel_t* out_pixel, unsigned rows, unsigned cols)
{
    pixel_channel_t* magnitude_v;
    pixel_channel_t_signed* deltaX_gray;
    pixel_channel_t_signed* deltaY_gray;

    hipMalloc((void**)&magnitude_v, sizeof(pixel_channel_t) * rows * cols);
    hipMalloc((void**)&deltaX_gray, sizeof(pixel_channel_t_signed) * rows * cols);
    hipMalloc((void**)&deltaY_gray, sizeof(pixel_channel_t_signed) * rows * cols);

    hipMemcpy(deltaX_gray, deltaX, rows * cols * sizeof(pixel_channel_t_signed), hipMemcpyHostToDevice);
    hipMemcpy(deltaY_gray, deltaY, rows * cols * sizeof(pixel_channel_t_signed), hipMemcpyHostToDevice);

    cu_magnitude << <(rows * cols) / 1024, 1024 >> > (deltaX_gray, deltaY_gray, magnitude_v, rows, cols);

    hipMemcpy(out_pixel, magnitude_v, rows * cols * sizeof(pixel_channel_t_signed), hipMemcpyDeviceToHost);

    hipFree(magnitude_v);
    hipFree(deltaX_gray);
    hipFree(deltaY_gray);
}

void cu_test_nonmax(pixel_channel_t* mag, pixel_channel_t_signed* deltaX, pixel_channel_t_signed* deltaY, pixel_channel_t* nms, unsigned rows, unsigned cols)
{
    pixel_channel_t* magnitude_v;
    pixel_channel_t* d_nms;
    pixel_channel_t_signed* deltaX_gray;
    pixel_channel_t_signed* deltaY_gray;

    hipMalloc((void**)&magnitude_v, sizeof(pixel_channel_t) * rows * cols);
    hipMalloc((void**)&d_nms, sizeof(pixel_channel_t) * rows * cols);
    hipMalloc((void**)&deltaX_gray, sizeof(pixel_channel_t_signed) * rows * cols);
    hipMalloc((void**)&deltaY_gray, sizeof(pixel_channel_t_signed) * rows * cols);

    hipMemcpy(magnitude_v, mag, rows * cols * sizeof(pixel_channel_t), hipMemcpyHostToDevice);
    hipMemcpy(deltaX_gray, deltaX, rows * cols * sizeof(pixel_channel_t_signed), hipMemcpyHostToDevice);
    hipMemcpy(deltaY_gray, deltaY, rows * cols * sizeof(pixel_channel_t_signed), hipMemcpyHostToDevice);

    cu_suppress_non_max << <(rows * cols) / 1024, 1024 >> > (magnitude_v, deltaX_gray, deltaY_gray, d_nms, rows, cols);

    hipMemcpy(nms, d_nms, rows * cols * sizeof(pixel_channel_t), hipMemcpyDeviceToHost);

    hipFree(magnitude_v);
    hipFree(d_nms);
    hipFree(deltaX_gray);
    hipFree(deltaY_gray);
}

void cu_test_hysteresis(pixel_channel_t* in, pixel_channel_t* out, unsigned rows, unsigned cols)
{
    pixel_channel_t* in_pixels, * out_pixels;
    unsigned* idx_map;

    /* allocate device memory */
    hipMalloc((void**)&in_pixels, rows * cols * sizeof(pixel_channel_t));
    hipMalloc((void**)&out_pixels, rows * cols * sizeof(pixel_channel_t));
    hipMalloc((void**)&idx_map, rows * cols * sizeof(idx_map[0]));

    /* copy original pixels to GPU device as in_pixels*/
    hipMemcpy(in_pixels, in, rows * cols * sizeof(pixel_channel_t), hipMemcpyHostToDevice);

    pixel_channel_t t_high = 0xFCC;
    pixel_channel_t t_low = 0x1FF;

    /* create task stream to sequence kernels */
    hipStream_t stream;
    hipStreamCreate(&stream);

    /* launch kernels */
    cu_hysteresis_high << <(rows * cols) / 1024, 1024, 0, stream >> > (out_pixels, in_pixels, idx_map, t_high, rows, cols);
    cu_hysteresis_low << <(rows * cols) / 1024, 1024, 0, stream >> > (out_pixels, in_pixels, idx_map, t_low, rows, cols);

    /* copy blurred pixels from GPU device back to host as out_pixels*/
    hipMemcpy(out, out_pixels, rows * cols * sizeof(pixel_channel_t), hipMemcpyDeviceToHost);

    hipFree(in_pixels);
    hipFree(out_pixels);
    hipFree(idx_map);
}

//*****************************************************************************************
// Entry point for serial program calling CUDA implementation
//*****************************************************************************************

void cu_detect_edges(pixel_channel_t* final_pixels, pixel_t* orig_pixels, int rows, int cols, double kernel[KERNEL_SIZE][KERNEL_SIZE])
{
    /* kernel execution configuration parameters */
    int num_blks = (rows * cols) / 1024;
    int thd_per_blk = 1024;
    int grid = 0;
    pixel_channel_t t_high = 0xFCC;
    pixel_channel_t t_low = 0xF5;

    /* device buffers */
    pixel_t* in, * out;
    pixel_channel_t* single_channel_buf0;
    pixel_channel_t* single_channel_buf1;
    pixel_channel_t_signed* deltaX;
    pixel_channel_t_signed* deltaY;
    double* d_blur_kernel;
    unsigned* idx_map;

    /* allocate device memory */
    hipMalloc((void**)&in, sizeof(pixel_t) * rows * cols);
    hipMalloc((void**)&out, sizeof(pixel_t) * rows * cols);
    hipMalloc((void**)&single_channel_buf0, sizeof(pixel_channel_t) * rows * cols);
    hipMalloc((void**)&single_channel_buf1, sizeof(pixel_channel_t) * rows * cols);
    hipMalloc((void**)&deltaX, sizeof(pixel_channel_t_signed) * rows * cols);
    hipMalloc((void**)&deltaY, sizeof(pixel_channel_t_signed) * rows * cols);
    hipMalloc((void**)&idx_map, sizeof(idx_map[0]) * rows * cols);
    hipMalloc((void**)&d_blur_kernel, sizeof(d_blur_kernel[0]) * KERNEL_SIZE * KERNEL_SIZE);

    /* data transfer image pixels to device */
    hipMemcpy(in, orig_pixels, rows * cols * sizeof(pixel_t), hipMemcpyHostToDevice);
    hipMemcpy(d_blur_kernel, kernel, sizeof(d_blur_kernel[0]) * KERNEL_SIZE * KERNEL_SIZE, hipMemcpyHostToDevice);

    /* run canny edge detection core - CUDA kernels */
    /* use streams to ensure the kernels are in the same task */
    hipStream_t stream;
    hipStreamCreate(&stream);
    cu_apply_gaussian_filter << <num_blks, thd_per_blk, grid, stream >> > (in, out, rows, cols, d_blur_kernel);
    cu_compute_intensity_gradient << <num_blks, thd_per_blk, grid, stream >> > (out, deltaX, deltaY, rows, cols);
    cu_magnitude << <num_blks, thd_per_blk, grid, stream >> > (deltaX, deltaY, single_channel_buf0, rows, cols);
    cu_suppress_non_max << <num_blks, thd_per_blk, grid, stream >> > (single_channel_buf0, deltaX, deltaY, single_channel_buf1, rows, cols);
    cu_hysteresis_high << <num_blks, thd_per_blk, grid, stream >> > (single_channel_buf0, single_channel_buf1, idx_map, t_high, rows, cols);
    cu_hysteresis_low << <num_blks, thd_per_blk, grid, stream >> > (single_channel_buf0, single_channel_buf1, idx_map, t_low, rows, cols);

    /* wait for everything to finish */
    hipDeviceSynchronize();

    /* copy result back to the host */
    hipMemcpy(final_pixels, single_channel_buf0, rows * cols * sizeof(pixel_channel_t), hipMemcpyDeviceToHost);

    /* cleanup */
    hipFree(in);
    hipFree(out);
    hipFree(single_channel_buf0);
    hipFree(single_channel_buf1);
    hipFree(deltaX);
    hipFree(deltaY);
    hipFree(idx_map);
    hipFree(d_blur_kernel);
}