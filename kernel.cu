
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "opencv2/highgui/highgui.hpp"
#include <opencv2/imgproc.hpp>
#include <iostream>
using namespace std;
using namespace cv;

void cpu_canny();

int main()
{
    cpu_canny();
    
    // Test to see if we can read BGR values
    /*
    Mat img;
    img = imread("Engine_in.PNG");
    int rows = img.rows;
    int col = img.cols;
    int depth = img.dims;
    cout << "Rows: " << rows << endl;
    cout << "Cols: " << rows << endl;
    cout << "Depth: " << depth << endl;
 
    for (int r = 0; r < 6; r++)
    {
        for (int c = 0; c < 6; c++)
        {
            Vec3b intensity = img.at<Vec3b>(r, c);
            cout << "Blue: " << (int)intensity.val[0] << " -- ";
            cout << "Green: " << (int)intensity.val[1] << " -- ";
            cout << "Red: " << (int)intensity.val[2] << " -- ";
            cout << endl;
        }
    }
    */
    return 0;
}

void cpu_canny()
{
    // openCV Matrices to read images
    Mat src, src_gray;
    Mat dst, detected_edges;

    // Canny variables
    int edgeThresh;
    int lowThresh = 100;
    int const max_lowThresh = 100;
    int ratio = 3;
    int kernel_size = 3;

    // Window Name
    char* window_name = "Edge Map";
    // Read image
    src = imread("Engine_in.PNG");
    if (!src.data)
    {
        cout << "No src picture found" << endl;
    }
    // Create matrix of same size and type
    dst.create(src.size(), src.type());
    // Covnert to grayscale
    cvtColor(src, src_gray, COLOR_BGR2GRAY);
    // Reduce noise (can experiment with size)
    blur(src_gray, detected_edges, Size(3, 3));
    // openCV Canny detecor
    Canny(detected_edges, detected_edges, lowThresh, lowThresh * ratio, kernel_size);
    
    // Explained as: "USING CANNY'S OUTPUT AS A MASK, DISPLAY RESULTS"
    //this fills dst with all 0s making it completely black
    dst = Scalar::all(0);

    // Copy src image to dst. copies only the pixelsin the locations where
    // they have non-zero vals. Since output of teh Canny detector is the edge
    // contours on a black background, the resulting dst will be black in all
    // the area but the detected edges
    src.copyTo(dst, detected_edges);
    // Display result
    imshow(window_name, dst);
    // Wait for key press.
    waitKey(0);
}