
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "opencv2/highgui/highgui.hpp"
#include <iostream>
using namespace std;
using namespace cv;

int main()
{
    Mat img;
    img = imread("Engine_in.PNG");
    int rows = img.rows;
    int col = img.cols;
    int depth = img.dims;
    cout << "Rows: " << rows << endl;
    cout << "Cols: " << rows << endl;
    cout << "Depth: " << depth << endl;
 
    for (int r = 0; r < 6; r++)
    {
        for (int c = 0; c < 6; c++)
        {
            Vec3b intensity = img.at<Vec3b>(r, c);
            cout << "Blue: " << (int)intensity.val[0] << " -- ";
            cout << "Green: " << (int)intensity.val[1] << " -- ";
            cout << "Red: " << (int)intensity.val[2] << " -- ";
            cout << endl;
        }
    }
    return 0;
}